#include "lapack_context_manager.h"

hipsolverHandle_t get_solver_handle()
{
  static hipsolverHandle_t handle;
  static int initialized = 0;

  if(!initialized)
  {
    hipsolverStatus_t stat;
    stat = hipsolverDnCreate(&handle);

    if (stat != HIPSOLVER_STATUS_SUCCESS)
    {
      printf("CUSOLVER initialization failed! Status: %d\n", stat);
      initialized = 0;
    }
    else
      initialized = 1;
  }
  return handle;
}
